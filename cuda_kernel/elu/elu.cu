#include "hip/hip_runtime.h"
##include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])


#define ELU_ALPHA 1.0f


// -------------------------------------- FP32 -------------------------------------- 
__device__ __forceinline__ float elu_f32(float x) {

    return (x > 0.0f) ? x : ELU_ALPHA * (expf(x) - 1.0f);

}

__device__ __forceinline__ half elu_half(half x) {
    half y = __hgt(x, __float2half(0.0f)) ? x :__hmul(_float2half(ELU_ALPHA),__hsub(hexp(x),__float2half(1.0f)));
    return y;
}


__global__ void elu_f32_kernel(float* x,float* y,int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] = elu_f32(x[idx]);
}

__global__ void elu_f32x4_kernel(float* x,float* y,int N){
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    if (idx < N) {
        float4 reg_x = FLOAT4(x[idx]);
        float4 reg_y;
        reg_y.x = elu_f32(reg_x.x);
        reg_y.y = elu_f32(reg_x.y);
        reg_y.z = elu_f32(reg_x.z);
        reg_y.w = elu_f32(reg_x.w);
        FLOAT4(y[idx]) = reg_y;
    }
}

// -------------------------------------- FP16 -------------------------------------- 
__global__ void elu_f16_kernel(half* x,half* y,int N){
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) y[idx] = elu_half(x[idx]);
}

__global__ void elu_f16x2_kernel(half* x,half* y,int N){
    int idx = 2 * (blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N) {
        half2 reg_x = HALF2(x[idx]);
        half2 reg_y;
        reg_y.x = elu_half(reg_x.x);
        reg_y.y = elu_half(reg_x.y);
        HALF2(y[idx]) = reg_y;
    }
}

__global__ void elu_f16x4_kernel(half* x,half* y,int N){
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 4;
    half2 reg_x1 = HALF2(x[idx]);
    half2 reg_x2 = HALF2(x[idx+2]);
    half2 reg_y1;
    half2 reg_y2;
    reg_y1.x = elu_half(reg_x1.x);
    reg_y1.y = elu_half(reg_x1.y);
    reg_y2.x = elu_half(reg_x2.x);
    reg_y2.y = elu_half(reg_x2.y);
    if((idx < N)) {HALF2(y[idx]= reg_y1); }
    if((idx+2 < N)) {HALF2(y[idx+2]= reg_y2);}
}


__global__ void elu_fp16x8_kernel(half* x,half* y,int N){
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    half2 reg_x1 = HALF2(x[idx]);
    half2 reg_x2 = HALF2(x[idx+2]);
    half2 reg_x3 = HALF2(x[idx+4]);
    half2 reg_x4 = HALF2(x[idx+6]);
    half2 reg_y1;
    half2 reg_y2;
    half2 reg_y3;
    half2 reg_y4;
    reg_y1.x = elu_half(reg_x1.x);
    reg_y1.y = elu_half(reg_x1.y);
    reg_y2.x = elu_half(reg_x2.x);
    reg_y2.y = elu_half(reg_x2.y);
    reg_y3.x = elu_half(reg_x3.x);
    reg_y3.y = elu_half(reg_x3.y);
    reg_y4.x = elu_half(reg_x4.x);
    reg_y4.y = elu_half(reg_x4.y);
    if((idx < N)) {HALF2(y[idx]= reg_y1); }
    if((idx+2 < N)) {HALF2(y[idx+2]= reg_y2);}
    if((idx+4 < N)) {HALF2(y[idx+4]= reg_y3);}
    if((idx+6 < N)) {HALF2(y[idx+6]= reg_y4);}
}


__global__ void elu_fp16x8_pack_kernel(half* x,half* y,int N){
    int idx = (blockIdx.x * blockDim.x + threadIdx.x) * 8;
    half pack_x[8];
    half pack_y[8];
    LDST128BITS(pack_x) = LDST128BITS(x[idx]);
    #pragma unroll
    for (int i = 0; i < 8; i++) {
        pack_y[i] = elu_half(pack_x[i]);
    }
    if((idx + 7) < N) {
        LDST128BITS(y[idx]) = LDST128BITS(pack_y[0]);
    }
}

void test_elu(){
    const int N = 1024;
    
    // 生成测试数据
    float *h_a = (float *)malloc(N * sizeof(float));
    half *h_a_f16 = (half *)malloc(N * sizeof(half));
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)rand() / RAND_MAX * 10.0f - 5.0f; // 生成-5到5之间的随机数
        h_a_f16[i] = __float2half(h_a[i]);
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 分配设备内存
    float *d_a, *d_y;
    half *d_a_f16, *d_y_f16;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_a_f16, N * sizeof(half));
    hipMalloc(&d_y_f16, N * sizeof(half));
    
    // 拷贝数据到设备
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_a_f16, h_a_f16, N * sizeof(half), hipMemcpyHostToDevice);

    // 测试 f32 kernel
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    float milliseconds = 0.0f;
    hipEventRecord(start);
    elu_f32_kernel<<<grid, block>>>(d_a, d_y, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f32 execution time: %f ms\n", milliseconds);

    // 测试 f32x4 kernel
    dim3 block_x4(64);  // 256/4
    dim3 grid_x4((N + block_x4.x * 4 - 1) / (block_x4.x * 4));
    hipEventRecord(start);
    elu_f32x4_kernel<<<grid_x4, block_x4>>>(d_a, d_y, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f32x4 execution time: %f ms\n", milliseconds);

    // 测试 f16 kernel
    hipEventRecord(start);
    elu_f16_kernel<<<grid, block>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f16 execution time: %f ms\n", milliseconds);

    // 测试 f16x2 kernel
    dim3 block_f16x2(128);  // 256/2
    dim3 grid_f16x2((N + block_f16x2.x * 2 - 1) / (block_f16x2.x * 2));
    hipEventRecord(start);
    elu_f16x2_kernel<<<grid_f16x2, block_f16x2>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f16x2 execution time: %f ms\n", milliseconds);

    // 测试 f16x4 kernel
    dim3 block_f16x4(64);  // 256/4
    dim3 grid_f16x4((N + block_f16x4.x * 4 - 1) / (block_f16x4.x * 4));
    hipEventRecord(start);
    elu_f16x4_kernel<<<grid_f16x4, block_f16x4>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f16x4 execution time: %f ms\n", milliseconds);

    // 测试 f16x8 kernel
    dim3 block_f16x8(32);  // 256/8
    dim3 grid_f16x8((N + block_f16x8.x * 8 - 1) / (block_f16x8.x * 8));
    hipEventRecord(start);
    elu_fp16x8_kernel<<<grid_f16x8, block_f16x8>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f16x8 execution time: %f ms\n", milliseconds);

    // 测试 f16x8 pack kernel
    hipEventRecord(start);
    elu_fp16x8_pack_kernel<<<grid_f16x8, block_f16x8>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("ELU kernel f16x8 pack execution time: %f ms\n", milliseconds);

    // 验证结果
    float *h_y = (float *)malloc(N * sizeof(float));
    half *h_y_f16 = (half *)malloc(N * sizeof(half));
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y_f16, d_y_f16, N * sizeof(half), hipMemcpyDeviceToHost);
    
    // 计算CPU结果
    float *h_y_cpu = (float *)malloc(N * sizeof(float));
    half *h_y_cpu_f16 = (half *)malloc(N * sizeof(half));
    for (int i = 0; i < N; i++) {
        h_y_cpu[i] = elu_f32(h_a[i]);
        h_y_cpu_f16[i] = elu_half(h_a_f16[i]);
    }
    
    // 比较结果
    float max_error = 0.0f;
    float max_error_f16 = 0.0f;
    for (int i = 0; i < N; i++) {
        float error = fabs(h_y[i] - h_y_cpu[i]);
        if (error > max_error) {
            max_error = error;
        }
        float error_f16 = fabs(__half2float(h_y_f16[i]) - __half2float(h_y_cpu_f16[i]));
        if (error_f16 > max_error_f16) {
            max_error_f16 = error_f16;
        }
    }
    printf("FP32 Max error: %f\n", max_error);
    printf("FP16 Max error: %f\n", max_error_f16);
    printf("ELU test passed!\n");
    
    // 释放资源
    free(h_a);
    free(h_a_f16);
    free(h_y);
    free(h_y_f16);
    free(h_y_cpu);
    free(h_y_cpu_f16);
    hipFree(d_a);
    hipFree(d_y);
    hipFree(d_a_f16);
    hipFree(d_y_f16);
}

int main(){
    test_elu();
    return 0;
}








#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>

#define WARP_SIZE 32
#define INT4(value) (reinterpret_cast<int4*>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4*>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2*>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162*>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4*>(&(value))[0])
#define MAX_EXP_F32  88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define MAX_EXP_F16 __float2half(11.089866488461016f)
#define MIN_EXP_F16 __float2half(-9.704060527839234f)
#define HALF_1 __float2half(1.0f)

#define THRESHOLD_MAX 3.0f
#define THRESHOLD_MIN -3.0f


// hardswish 公式 if x=< -3 返回0 if x>=3 返回x else 返回x*(x+3)/6

__device__ __forceinline__ float hardswish(float x) {

    if (x >= THRESHOLD_MAX){
        return x;
    } else if (x <= THRESHOLD_MIN){
        return 0.0f; 
    } else {
        return x * (x + 3.0f) / 6.0f;
    }
}


__device__ __forceinline__ half hardswish_half(half x) {
    if (x >= __float2half(THRESHOLD_MAX)){
        return x;
    } else if (x <= __float2half(THRESHOLD_MIN)){
        return __float2half(0.0f); 
    } else {
        return x * (x + __float2half(3.0f)) / __float2half(6.0f);
    }
}

__global__ void hardswish_fp32_kernel(float x, float* y,int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = hardswish(x);
    }
}

__global__ void hardswish_fp32x4_kernel(float *x, float* y,int N) {
    int idx = 4*(blockIdx.x * blockDim.x + threadIdx.x);
    if (idx < N) {
        FLOAT4 reg_x = FLOAT4(x[idx]);
        FLOAT4 reg_y;
        reg_y.x = hardswish(reg_x.x);
        reg_y.y = hardswish(reg_x.y);
        reg_y.z = hardswish(reg_x.z);
        reg_y.w = hardswish(reg_x.w);
        FLOAT4(y[idx]) = reg_y;
    }
}

__global__ void hardswish_fp16_kernel(half *x, half* y,int N) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < N) {
        y[idx] = hardswish_half(x[idx]);
    }
}

__global__ void hardswish_fp16x2_kernel(half *x, half* y,int N) {
    int idx = 2*(blockIdx.x * blockDim.x + threadIdx.x);
    HALF2 reg_x = HALF2(x[idx]);
    HALF2 reg_y;
    reg_y.x = hardswish_half(reg_x.x);
    reg_y.y = hardswish_half(reg_x.y);
    HALF2(y[idx]) = reg_y;
}


__global__ void hardswish_fp16x8_kernel(half *x, half* y,int N) {
   int idx = 8*(blockIdx.x * blockDim.x + threadIdx.x);
   half2 reg_x_0 = HALF2(x[idx]);
   half2 reg_x_1 = HALF2(x[idx+2]);
   half2 reg_x_2 = HALF2(x[idx+4]);
   half2 reg_x_3 = HALF2(x[idx+6]);

   half2 reg_y_0, reg_y_1, reg_y_2, reg_y_3;
   reg_y_0.x = hardswish_half(reg_x_0.x);
   reg_y_0.y = hardswish_half(reg_x_0.y);
   reg_y_1.x = hardswish_half(reg_x_1.x);
   reg_y_1.y = hardswish_half(reg_x_1.y);
   reg_y_2.x = hardswish_half(reg_x_2.x);
   reg_y_2.y = hardswish_half(reg_x_2.y);
   reg_y_3.x = hardswish_half(reg_x_3.x);
   reg_y_3.y = hardswish_half(reg_x_3.y);
   if ((idx) < N) {
        HALF2(y[idx]) = reg_y_0;
   }
   if ((idx+2) < N) {
        HALF2(y[idx+2]) = reg_y_1;
   }
   if ((idx+4) < N) {
        HALF2(y[idx+4]) = reg_y_2;
  
    }
   if ((idx+6) < N) {
        HALF2(y[idx+6]) = reg_y_3; 
   }
}

__global__ void hardswish_fp16x8_pack_kernel(half *x, half* y,int N) {
   int idx = 8*(blockIdx.x * blockDim.x + threadIdx.x);
   half2 reg_x_pack[8], reg_y_pack[8];

    LDST128BITS(reg_x_pack[0]) = LDST128BITS(x[idx]);
    #pragma unroll
    for (int i = 0; i < 8; i++) {
        reg_y_pack[i] = hardswish_half(reg_x_pack[i]);
    }

    if ((idx + 7) < N) {
        LDST128BITS(y[idx]) = LDST128BITS(reg_y_pack[0]);
    }
}

void test_hardswish() {
    const int N = 1024;
    
    // 生成测试数据
    float *h_a = (float *)malloc(N * sizeof(float));
    half *h_a_f16 = (half *)malloc(N * sizeof(half));
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)rand() / RAND_MAX * 10.0f - 5.0f; // 生成-5到5之间的随机数
        h_a_f16[i] = __float2half(h_a[i]);
    }
    
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 分配设备内存
    float *d_a, *d_y;
    half *d_a_f16, *d_y_f16;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    hipMalloc(&d_a_f16, N * sizeof(half));
    hipMalloc(&d_y_f16, N * sizeof(half));
    
    // 拷贝数据到设备
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_a_f16, h_a_f16, N * sizeof(half), hipMemcpyHostToDevice);

    // 测试 fp32 kernel
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    float milliseconds = 0.0f;
    hipEventRecord(start);
    hardswish_fp32_kernel<<<grid, block>>>(d_a, d_y, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp32 execution time: %f ms\n", milliseconds);

    // 测试 fp32x4 kernel
    dim3 block_x4(64);  // 256/4
    dim3 grid_x4((N + block_x4.x * 4 - 1) / (block_x4.x * 4));
    hipEventRecord(start);
    hardswish_fp32x4_kernel<<<grid_x4, block_x4>>>(d_a, d_y, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp32x4 execution time: %f ms\n", milliseconds);

    // 测试 fp16 kernel
    hipEventRecord(start);
    hardswish_fp16_kernel<<<grid, block>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp16 execution time: %f ms\n", milliseconds);

    // 测试 fp16x2 kernel
    dim3 block_f16x2(128);  // 256/2
    dim3 grid_f16x2((N + block_f16x2.x * 2 - 1) / (block_f16x2.x * 2));
    hipEventRecord(start);
    hardswish_fp16x2_kernel<<<grid_f16x2, block_f16x2>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp16x2 execution time: %f ms\n", milliseconds);

    // 测试 fp16x8 kernel
    dim3 block_f16x8(32);  // 256/8
    dim3 grid_f16x8((N + block_f16x8.x * 8 - 1) / (block_f16x8.x * 8));
    hipEventRecord(start);
    hardswish_fp16x8_kernel<<<grid_f16x8, block_f16x8>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp16x8 execution time: %f ms\n", milliseconds);

    // 测试 fp16x8 pack kernel
    hipEventRecord(start);
    hardswish_fp16x8_pack_kernel<<<grid_f16x8, block_f16x8>>>(d_a_f16, d_y_f16, N);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("HardSwish kernel fp16x8 pack execution time: %f ms\n", milliseconds);

    // 验证结果
    float *h_y = (float *)malloc(N * sizeof(float));
    half *h_y_f16 = (half *)malloc(N * sizeof(half));
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
    hipMemcpy(h_y_f16, d_y_f16, N * sizeof(half), hipMemcpyDeviceToHost);
    
    // 计算CPU结果
    float *h_y_cpu = (float *)malloc(N * sizeof(float));
    half *h_y_cpu_f16 = (half *)malloc(N * sizeof(half));
    for (int i = 0; i < N; i++) {
        // 使用标准HardSwish函数计算CPU结果
        float x = h_a[i];
        if (x <= -3.0f) {
            h_y_cpu[i] = 0.0f;
        } else if (x >= 3.0f) {
            h_y_cpu[i] = x;
        } else {
            h_y_cpu[i] = x * (x + 3.0f) / 6.0f;
        }
        h_y_cpu_f16[i] = __float2half(h_y_cpu[i]);
    }
    
    // 比较结果
    float max_error = 0.0f;
    float max_error_f16 = 0.0f;
    for (int i = 0; i < N; i++) {
        float error = fabs(h_y[i] - h_y_cpu[i]);
        if (error > max_error) {
            max_error = error;
        }
        float error_f16 = fabs(__half2float(h_y_f16[i]) - __half2float(h_y_cpu_f16[i]));
        if (error_f16 > max_error_f16) {
            max_error_f16 = error_f16;
        }
    }
    printf("FP32 Max error: %f\n", max_error);
    printf("FP16 Max error: %f\n", max_error_f16);
    printf("HardSwish test passed!\n");
    
    // 释放资源
    free(h_a);
    free(h_a_f16);
    free(h_y);
    free(h_y_f16);
    free(h_y_cpu);
    free(h_y_cpu_f16);
    hipFree(d_a);
    hipFree(d_y);
    hipFree(d_a_f16);
    hipFree(d_y_f16);
    
    hipEventDestroy(start);
    hipEventDestroy(stop);
}

int main() {
    test_hardswish();
    return 0;
}
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <float.h>
#include <vector>
#include <algorithm>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_bf16.h>
#include <hip/hip_fp8.h>


#define WARP_SIZE 256
#define WARP_SIZE_S 16
#define PAD 1
#define INT4(value) (reinterpret_cast<int4 *>(&(value))[0])
#define FLOAT4(value) (reinterpret_cast<float4 *>(&(value))[0])
#define HALF2(value) (reinterpret_cast<half2 *>(&(value))[0])
#define BFLOAT2(value) (reinterpret_cast<__hip_bfloat162 *>(&(value))[0])
#define LDST128BITS(value) (reinterpret_cast<float4 *>(&(value))[0])
#define MAX_EXP_F32 88.3762626647949f
#define MIN_EXP_F32 -88.3762626647949f
#define MAX_EXP_F16 __float2half(11.089866488461016f)
#define MIN_EXP_F16 __float2half(-9.704060527839234f)

// col2row means read x[row][col] and write y[col][row] // 判断连续内存
// row2col means read x[col][row] and write y[row][col]
//  1 2 3               1  4 
//  4 5 6       -- >    2  5       连续内存 x 为 1，2，3，4，5，6 
//                      3  6
__global__ void matrix_transpose_fp32_col2row_kernel(float *x,float *y,int row,int col) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int g_row = tid / col;
    const int g_col = tid % col;
    if (tid < row * col) {
        y[g_col * row + g_row] = x[tid];
    }
}

__global__ void matrix_transpose_fp32_row2col_kernel(float *x,float *y,int row,int col) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int g_col = tid / row;
    const int g_row = tid % row;
    if (tid < row * col) {
        y[tid] = x[g_row * col + g_col];
    }
}

__global__ void matrix_transpose_fp32x4_col2row_kernel(float *x,float *y,int row,int col) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int g_row = (tid * 4) / col;
    const int g_col = (tid*4) % col; 
    if ( g_row < row && (g_col+3) < col) {
        float4 reg_x = FLOAT4(x[tid])
        y[g_col * row + g_row] = reg_x.x;
        y[(g_col+1) * row + g_row] = reg_x.y;
        y[(g_col+1) * row + g_row ] = reg_x.z;
        y[(g_col+1) * row + g_row] = reg_x.w;
    }
}

__global__ void matrix_transpose_fp32x4_row2col_kernel(float *x,float *y,int row,int col) {
    const int tid = threadIdx.x + blockIdx.x * blockDim.x;
    const int g_col = (tid * 4) / row;
    const int g_row = (tid*4) % row;
    if ( g_col < col && (g_row+3) < row) {
        float4 reg_y;
        reg_y.x = x[g_row * col + g_col];
        reg_y.y = x[(g_row+1) * col + g_col];
        reg_y.z = x[(g_row+2) * col + g_col];
        reg_y.w = x[(g_row+3) * col + g_col];
        FLOAT4(y[tid]) = reg_y;
    }
}


__global__ void matrix_transpose_fp32_diag2d_kernel(float *x,float *y,int row,int col){
    const int block_y = blockIdx.x;
    const int block_x = (blockIdx.y + blockIdx.x) % gridDim.x;
    const int global_col = block_x * blockDim.x + threadIdx.x;
    const int global_row = block_y * blockDim.y + threadIdx.y;
//   const int block_y = blockIdx.x;
//   const int block_x = (blockIdx.x + blockIdx.y) % gridDim.x;
//   const int global_col = threadIdx.x + blockDim.x * block_x;
//   const int global_row = threadIdx.y + blockDim.y * block_y;

    if (global_col < col && global_row < row){
        y[global_row * col + global_col] = x[global_col * row + global_row];
    }
}

__global__ void mat_transpose_f32_col2row2d_kernel(float *x, float *y, const int row, const int col) {
  const int global_x = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_y = blockIdx.y * blockDim.y + threadIdx.y;
  if (global_x < col && global_y < row) {
    y[global_x * row + global_y] = x[global_y * col + global_x];
  }
}

__global__ void mat_transpose_f32_row2col2d_kernel(float *x, float *y, const int row, const int col) {
    // row2col2d 等价于 col2row2d
  const int global_y = blockIdx.x * blockDim.x + threadIdx.x;
  const int global_x = blockIdx.y * blockDim.y + threadIdx.y;
  if (global_y < col && global_x < row) {
    y[global_y * row + global_x] = x[global_x * col + global_y];
  }
}



void test_matrix_transpose() {
    const int ROW = 32;
    const int COL = 32;
    const int N = ROW * COL;
    
    // 生成测试数据
    float *h_a = (float *)malloc(N * sizeof(float));
    for (int i = 0; i < N; i++) {
        h_a[i] = (float)rand() / RAND_MAX * 10.0f - 5.0f; // 生成-5到5之间的随机数
    }

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    
    // 分配设备内存
    float *d_a, *d_y;
    hipMalloc(&d_a, N * sizeof(float));
    hipMalloc(&d_y, N * sizeof(float));
    
    // 拷贝数据到设备
    hipMemcpy(d_a, h_a, N * sizeof(float), hipMemcpyHostToDevice);

    // 调用核函数 - col2row
    dim3 block(256);
    dim3 grid((N + block.x - 1) / block.x);
    float milliseconds = 0.0f;
    hipEventRecord(start);
    matrix_transpose_fp32_col2row_kernel<<<grid, block>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose col2row kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - row2col
    milliseconds = 0.0f;
    hipEventRecord(start);
    matrix_transpose_fp32_row2col_kernel<<<grid, block>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose row2col kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - col2row x4
    dim3 block4(64);
    dim3 grid4((N + block.x*4 - 1) / block.x*4);
    milliseconds = 0.0f;
    hipEventRecord(start);
    matrix_transpose_fp32x4_col2row_kernel<<<grid4, block4>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose col2row x4 kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - row2col x4
    milliseconds = 0.0f;
    hipEventRecord(start);
    matrix_transpose_fp32x4_row2col_kernel<<<grid4, block4>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose row2col x4 kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - diag2d
    dim3 block2d(16, 16);
    dim3 grid2d((COL + block2d.x - 1) / block2d.x, (ROW + block2d.y - 1) / block2d.y);
    milliseconds = 0.0f;
    hipEventRecord(start);
    matrix_transpose_fp32_diag2d_kernel<<<grid2d, block2d>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose diag2d kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - col2row2d
    milliseconds = 0.0f;
    hipEventRecord(start);
    mat_transpose_f32_col2row2d_kernel<<<grid2d, block2d>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose col2row2d kernel execution time: %f ms\n", milliseconds);

    // 调用核函数 - row2col2d
    milliseconds = 0.0f;
    hipEventRecord(start);
    mat_transpose_f32_row2col2d_kernel<<<grid2d, block2d>>>(d_a, d_y, ROW, COL);
    hipEventRecord(stop);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&milliseconds, start, stop);
    printf("Matrix transpose row2col2d kernel execution time: %f ms\n", milliseconds);

    // 验证结果
    float *h_y = (float *)malloc(N * sizeof(float));
    hipMemcpy(h_y, d_y, N * sizeof(float), hipMemcpyDeviceToHost);
    
    // 计算CPU结果
    float *h_y_cpu = (float *)malloc(N * sizeof(float));
    for (int i = 0; i < ROW; i++) {
        for (int j = 0; j < COL; j++) {
            h_y_cpu[j * ROW + i] = h_a[i * COL + j];
        }
    }
    
    // 比较结果
    float max_error = 0.0f;
    for (int i = 0; i < N; i++) {
        float error = fabs(h_y[i] - h_y_cpu[i]);
        if (error > max_error) {
            max_error = error;
        }
    }
    printf("Max error: %f\n", max_error);
    printf("Matrix transpose test passed!\n");
    
    // 释放资源
    free(h_a);
    free(h_y);
    free(h_y_cpu);
    hipFree(d_a);
    hipFree(d_y);
}

int main() {
    test_matrix_transpose();
    return 0;
}
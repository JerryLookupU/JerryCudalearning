#include <stdio.h>
#include <hip/hip_runtime.h>



__forceinline__ __device__ int warp_reduce_sum(int value) {
    value += __shfl_xor_sync(0xffffffff, value, 16);
    value += __shfl_xor_sync(0xffffffff, value, 8);
    value += __shfl_xor_sync(0xffffffff, value, 4);
    value += __shfl_xor_sync(0xffffffff, value, 2);
    value += __shfl_xor_sync(0xffffffff, value, 1);
    return value;
}

__forceinline__ __device__ float half_warp_reduce_max(float value) {
    auto mask = __activemask();
    // The mask be in `{0xffffffff, 0xffff}`
    value = max(value, __shfl_xor_sync(mask, value, 8));
    value = max(value, __shfl_xor_sync(mask, value, 4));
    value = max(value, __shfl_xor_sync(mask, value, 2));
    value = max(value, __shfl_xor_sync(mask, value, 1));
    return value;
}

__global__ void test_warp_reduce_sum(int* input, int* output) {
    int tid = threadIdx.x;
    output[tid] = warp_reduce_sum(input[tid]);
    // 一个warp 通常32个线程
}

__global__ void test_half_warp_reduce_max(float* input, float* output) {
    int tid = threadIdx.x;
    output[tid] = half_warp_reduce_max(input[tid]);
}

void run_tests() {
    const int size = 32;
    int* h_input = new int[size];
    int* h_output = new int[size];
    float* h_finput = new float[size];
    float* h_foutput = new float[size];
    
    // 测试warp_reduce_sum
    for (int i = 0; i < size; ++i) h_input[i] = i + 1;
    
    int *d_input, *d_output;
    float *d_finput, *d_foutput;
    hipMalloc(&d_input, size * sizeof(int));
    hipMalloc(&d_output, size * sizeof(int));
    hipMemcpy(d_input, h_input, size * sizeof(int), hipMemcpyHostToDevice);
    
    test_warp_reduce_sum<<<1, size>>>(d_input, d_output);
    hipMemcpy(h_output, d_output, size * sizeof(int), hipMemcpyDeviceToHost);
    
    // 验证结果
    int expected_sum = (size * (size + 1)) / 2;
    for (int i = 0; i < size; ++i) {
        if (h_output[i] != expected_sum) {
            printf("warp_reduce_sum test failed at thread %d: got %d, expected %d\n", 
                   i, h_output[i], expected_sum);
        }
    }
    
    // 测试half_warp_reduce_max
    for (int i = 0; i < size; ++i) h_finput[i] = (float)(i + 1);
    
    hipMalloc(&d_finput, size * sizeof(float));
    hipMalloc(&d_foutput, size * sizeof(float));
    hipMemcpy(d_finput, h_finput, size * sizeof(float), hipMemcpyHostToDevice);
    
    test_half_warp_reduce_max<<<1, size>>>(d_finput, d_foutput);
    hipMemcpy(h_foutput, d_foutput, size * sizeof(float), hipMemcpyDeviceToHost);
    
    // 验证结果 - only check first 16 threads since it's a half-warp operation
    float expected_max = (float)16;
    for (int i = 0; i < 16; ++i) {
        if (h_foutput[i] != expected_max) {
            printf("half_warp_reduce_max test failed at thread %d: got %f, expected %f\n", 
                   i, h_foutput[i], expected_max);
        }
    }
    
    // 清理
    delete[] h_input;
    delete[] h_output;
    delete[] h_finput;
    delete[] h_foutput;
    hipFree(d_input);
    hipFree(d_output);
    hipFree(d_finput);
    hipFree(d_foutput);
}

int main() {
    run_tests();
    printf("All tests completed.\n");
    return 0;
}
